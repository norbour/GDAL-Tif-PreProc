#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>	
#include <math.h>
 
#include <hip/hip_runtime.h>
#include <>

#include "../include/book.h"
#include "../include/commCuda.h"
#include "../../include/tiffImageIO.h"

/**
 * @Device
 * Active factor GeoTiff pixel normalization.
 */
__global__ void normalizeActiveRasterPixel( float  *pixelMatrix, 
	                                        int    *nPixels, 
										    double *rasterMinMax ) 
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < *nPixels)
	{
		
		if ( pixelMatrix[tid] - 0xE0000000 != 0 ) // float nullPixelValue = 0xE0000000;
		{
			pixelMatrix[tid] = (pixelMatrix[tid] - rasterMinMax[0]) /
				               (rasterMinMax[1]  - rasterMinMax[0]);
		}
		
		tid += blockDim.x * gridDim.x; 
	}
}

/**
 * @Device
 * Negative factor GeoTiff pixel normalization.
 */
__global__ void normalizeNegativeRasterPixel( float  *pixelMatrix, 
	                                          int    *nPixels, 
	                                          double *rasterMinMax ) 
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < *nPixels)
	{
		pixelMatrix[tid] = (rasterMinMax[1] - pixelMatrix[tid]) /
			               (rasterMinMax[1] - rasterMinMax[0]);

		tid += blockDim.x * gridDim.x; 
	}
}

/**
 * <Core Function>
 * Factor GeoTiff pixel normalization.
 * @param pixelMatrix  -> raster pixel value array
 * @param tiffWidth    -> raster width
 * @param tiffHeigth   -> raster length
 * @param rasterMinMax -> min & max value in raster pixels
 * @param factorType   -> evaluation factor type (Active/Negative)
 */
void rasterPixelNormalization(float         *pixelMatrix, 
	                          int           tiffWidth, 
							  int           tiffHeigth, 
					          const double  rasterMinMax[2],
                              envFactorType factorType) 
{
	int nPixels = tiffWidth * tiffHeigth;

	float  *dev_pixelMatrix  = NULL;    
	int    *dev_nPixels      = NULL;    
	double *dev_rasterMinMax = NULL;     

	HANDLE_ERROR( hipMalloc( (void**)&dev_pixelMatrix, nPixels * sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_nPixels,               sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_rasterMinMax,      2 * sizeof(double) ) );

	HANDLE_ERROR( hipMemcpy( dev_pixelMatrix,
		                      pixelMatrix,
							  nPixels * sizeof(float),
							  hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_nPixels,
		                      &nPixels,
							  sizeof(int),
							  hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_rasterMinMax,
		                      rasterMinMax,
							  2 * sizeof(double),
							  hipMemcpyHostToDevice ) );

	/******************** Preparation for CUDA execution time recording ********************/

	hipEvent_t timeStartEvent, timeEndEvent;

	HANDLE_ERROR( hipEventCreate( &timeStartEvent, 0 ) );
	HANDLE_ERROR( hipEventCreate( &timeEndEvent, 0 ) );

	HANDLE_ERROR( hipEventRecord( timeStartEvent, 0 ) );

	/******************** ******************************************** ********************/

	if ( factorType == factor_Active ) 
	{
		normalizeActiveRasterPixel<<<128, 128>>>( dev_pixelMatrix, 
			                                      dev_nPixels, 
			                                      dev_rasterMinMax );
	}
	else
	{
		normalizeNegativeRasterPixel<<<128, 128>>>( dev_pixelMatrix, 
			                                        dev_nPixels, 
			                                        dev_rasterMinMax );
	}

	/********************** Check out CUDA execution time recording ***********************/
	HANDLE_ERROR( hipEventRecord( timeEndEvent, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(timeEndEvent) );

	float elapsedTime = 0;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, timeStartEvent, timeEndEvent ) );

	printf( "Time Consumption: %f ms. \n", elapsedTime );

	HANDLE_ERROR( hipEventDestroy( timeStartEvent ) );
	HANDLE_ERROR( hipEventDestroy( timeEndEvent ) );
	/******************** ******************************************** ********************/

	HANDLE_ERROR( hipMemcpy( pixelMatrix,
		                      dev_pixelMatrix,
		                      nPixels * sizeof(float),
		                      hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipFree( dev_pixelMatrix ) );
	HANDLE_ERROR( hipFree( dev_nPixels ) );
	HANDLE_ERROR( hipFree( dev_rasterMinMax ) );
}

/**
 * <Interface>
 * Factor GeoTiff pixel normalization.
 * @param srcTifFile         ->  Source GeoTiff file path
 * @param outputTifFile      ->  Result output file path
 * @param factorType         ->  Evaluation factor type (Active/Negative)
 */
void geoTiffRasterPixelNormalization( const char    srcTifFile[],
	                                  const char    outputTifFile[],
                                      envFactorType factorType )
{
	float  *rasterPixels = NULL;
	int    tifWidth, tifLength;
	double *rasterMinMax;

	rasterMinMax = (double*)malloc( sizeof(double) * 2 );
	if ( rasterMinMax == NULL )
	{
		ERROR_INFO( "Out of memory" );
		return;
	}

	readTiffImageToMatrix( srcTifFile, 
		                   1, 
						   &rasterPixels );

	getTiffWidthLength( srcTifFile,
		                1,
						&tifWidth,
						&tifLength );

	getTiffMinMax( srcTifFile,
		           1,
				   rasterMinMax,
				   1 );

	rasterPixelNormalization( rasterPixels, 
	                          tifWidth, 
							  tifLength, 
					          rasterMinMax,
                              factorType );

	writeTiffImageRefSrc( outputTifFile, 
		                  srcTifFile, 
		                  1, 
		                  rasterPixels );

	double normalizedRasterMinMax[] = { 0, 1 };

	alterRasterMinMax( outputTifFile,
		               1,
	                   normalizedRasterMinMax );

	free(rasterMinMax);
}