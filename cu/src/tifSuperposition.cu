#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>	
#include <math.h>

#include <hip/hip_runtime.h>
#include <>

#include "../include/book.h"
#include "../include/commonCudaHeader.h"
#include "../../include/tiffImageIO.h"

/**
 * @Device
 * Active factor GeoTiff pixel normalization.
 */
__global__ void superposeTifRasters( char   *factorTifNames[], 
	                                 double *nPixels, 
								     int    *factorNum ) 
{

}